#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <random>
#include <limits>
#include <assert.h>
#include <cmath>
#include "sparse_matrix.cuh"

constexpr double EPSILON = 10e-12;

int main(int argc, char** argv)
{

  std::vector<std::string> input_matrices = {"../data/jgl009.mtx"};
  std::vector<std::string> output_vectors = {"../data/jgl009_id.mtx"};
  for (size_t i = 0; i < input_matrices.size(); ++i)
  {
    std::cout << "Reading: " << input_matrices[i] << std::endl;;
    HostCSRMatrix host_csr(input_matrices[i]);
    std::cout << "Reading: " << output_vectors[i] << std::endl;;
    HostCSRMatrix host_csr_result(output_vectors[i]);
    std::cout << "Read matrices." << std::endl;

    assert(host_csr.num_rows() == host_csr.num_cols());
    assert(host_csr.num_cols() == host_csr_result.num_rows());
    assert(host_csr_result.num_cols() == 1);
    std::cout << "Passed asserts. Rehydrating." << std::endl;
    thrust::host_vector<double> host_result = host_csr_result.rehydrate();
    std::cout << "Rehydrated." << std::endl;
    const size_t n = host_csr.num_rows();

    auto dev_csr = host_to_dev(host_csr);
    
    thrust::device_vector<double> id_vector(n);
    thrust::device_vector<double> out_vector(n);
    thrust::fill(id_vector.begin(), id_vector.end(), 1);

    dev_csr.matvec(id_vector, out_vector);
    for (size_t i = 0; i < n; ++i)
    {
      if (abs(out_vector[i] - host_result[i]) > EPSILON)
      {
        std::cout << i << ": " << host_result[i] << ", " << out_vector[i] << std::endl;
      }
    }
  }
}
