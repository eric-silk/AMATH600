#include "hip/hip_runtime.h"
#include "sparse_matrix.cuh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <random>
#include <limits>

int main(int argc, char** argv)
{
  size_t n;
  if (argc >= 2)
  {
    n = atoi(argv[1]);
  }
  else
  {
    n = 6;
  }

  HostCSRMatrix host_csr(n, n);
  
  // Generate N random numbers and n i,j coords
  std::random_device rd;
  std::mt19937 gen(rd());
  // The distribution is inclusive, hence the n-1
  std::uniform_int_distribution<size_t> coord_distr(0, n-1);
  const double double_min = std::numeric_limits<double>::min();
  const double double_max = std::numeric_limits<double>::max();
  std::uniform_real_distribution<double> val_distr(double_min, double_max);

  host_csr.open_for_pushback();
  for (size_t i = 0; i < n; ++i)
  {
    size_t row = coord_distr(gen);
    size_t col = coord_distr(gen);
    double value = val_distr(gen);
    host_csr.push_back(row, col, value);
  }
  host_csr.close_for_pushback();

  auto dev_csr = host_to_dev(host_csr);
}
