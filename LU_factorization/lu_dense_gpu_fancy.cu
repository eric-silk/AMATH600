#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <math.h>
#include <assert.h>

#include "strided_iterator.cuh"

struct LU
{
  LU() = default;
  std::vector<double> L;
  std::vector<double> U;
  size_t n;
};

// Borrowed from the transformations example code
// https://docs.nvidia.com/cuda/thrust/index.html#transformations
struct daxpy_functor
{
  const double a;
  daxpy_functor(double _a) : a(_a) {}
  __host__ __device__
    double operator()(const double& x, const double&y) const
    {
      return a * x + y;
    }
};

void daxpy(double A, thrust::device_vector<double>& X, thrust::device_vector<double>& Y)
{
  thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), daxpy_functor(A));
}

struct to_zero_functor
{
  const double epsilon;
  to_zero_functor(double _epsilon) : epsilon(_epsilon) {}
  __host__ __device__
    double operator()(const double& x) const
    {
      if (std::abs(x) <= epsilon)
        return 0;
      else
        return x;
    }
};

void to_zero(double epsilon, thrust::device_vector<double>& X)
{
  thrust::transform(X.begin(), X.end(), X.begin(), to_zero_functor(epsilon));
}

LU LU_factorization(const std::vector<double>& A, size_t n);
void print_matrix(const std::vector<double>& A, size_t n);
double rand_0_1(void);

int main(int argc, char **argv)
{
  int n = 0;
  if (argc >= 2)
  {
    n = atoi(argv[1]);
  }
  else
  {
    n = 1024;
  }

  std::vector<double> A;
  A.resize(n*n);
  std::srand(std::time(nullptr));
  std::generate(A.begin(), A.end(), rand_0_1);

  auto factored = LU_factorization(A, n);

  return 0;
}

double rand_0_1(void)
{
  return ((double) rand() / (RAND_MAX));
}

void print_matrix(const std::vector<double>& A, size_t n)
{
  for (size_t row = 0; row < n; ++row)
  {
    for (size_t col = 0; col < n; ++col)
    {
      std::cout << A[n*row+col] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

LU LU_factorization(const std::vector<double>& A, const size_t n)
{
  assert(A.size() == n*n);
  std::vector<double> U(n*n);
  std::vector<double> L(n*n, 0);

  thrust::host_vector<double>   U_h = A;
  thrust::device_vector<double> U_d = U_h;
  thrust::device_vector<double> Coeffs(n);

  // Let's start with just iterating manually over columns
  // Probably replace this with a counting iterator
  for (size_t col = 0; col < n; ++col)
  {
    // Constant iterator for the current top row
    thrust::constant_iterator<double> numerator(U_d[col*(n+1)]);
    // strided iterator for the coff calcs
    strided_range denominator(U_d.begin()+(n*col)+n, U.end(), n);
    // Coeff iterator
    auto first = thrust::make_zip_iterator(thrust::make_tuple(numerator, denominator.begin()));
    auto last = thrust::make_zip_iterator(thrust::make_tuple(numerator, denominator.end()));

    thrust::copy(first, last, std::ostream_iterator<double>(std::cout, "\n"));
  }

  LU retval;
  retval.U = U;
  retval.L = L;

  return retval;
}
