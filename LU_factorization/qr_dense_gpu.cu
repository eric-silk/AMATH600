#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <math.h>
#include <assert.h>

// http://www.seas.ucla.edu/~vandenbe/133A/lectures/qr.pdf
// https://www.math.ucla.edu/~yanovsky/Teaching/Math151B/handouts/GramSchmidt.pdf

struct QR
{
  QR
  std::vector<double> Q;
  std::vector<double> R;
  size_t n;
};

// Borrowed from the transformations example code
// https://docs.nvidia.com/cuda/thrust/index.html#transformations
struct daxpy_functor
{
  const double a;
  daxpy_functor(double _a) : a(_a) {}
  __host__ __device__
    double operator()(const double& x, const double&y) const
    {
      return a * x + y;
    }
};

void daxpy(double A, thrust::device_vector<double>& X, thrust::device_vector<double>& Y)
{
  thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), daxpy_functor(A));
}

struct to_zero_functor
{
  const double epsilon;
  to_zero_functor(double _epsilon) : epsilon(_epsilon) {}
  __host__ __device__
    double operator()(const double& x) const
    {
      if (std::abs(x) <= epsilon)
        return 0;
      else
        return x;
    }
};

void to_zero(double epsilon, thrust::device_vector<double>& X)
{
  thrust::transform(X.begin(), X.end(), X.begin(), to_zero_functor(epsilon));
}

QR QR(const std::vector<double>& A, size_t n);
void print_matrix(const std::vector<double>& A, size_t n);
double rand_0_1(void);

int main(int argc, char **argv)
{
  int n = 0;
  if (argc >= 2)
  {
    n = atoi(argv[1]);
  }
  else
  {
    n = 1024;
  }

  std::vector<double> A;
  A.resize(n*n);
  std::srand(std::time(nullptr));
  std::generate(A.begin(), A.end(), rand_0_1);

  auto factored = QR_factorization(A, n);

  return 0;
}

double rand_0_1(void)
{
  return ((double) rand() / (RAND_MAX));
}

void print_matrix(const std::vector<double>& A, size_t n)
{
  for (size_t row = 0; row < n; ++row)
  {
    for (size_t col = 0; col < n; ++col)
    {
      std::cout << A[n*row+col] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

// I've opted to work with CSR's for now; CSC's may be more well suited
// but bleh
std::vector<double> get_column_of_row_major_cpu(std::vector<double>& A, const size_t n, const size_t colnumber)
{
  std::vector<double> column(n);
  for (size_t i = 0; i < n; ++i)
  {
    column[n] = A[i*n+colnumber];
  }

  return column;
}

thrust::device_vector<double> get_column_of_row_major_gpu(thrust::device_vector<double> matrix,
                                                          const size_t n,
                                                          const size_t colnumber)
{
  // Gather up the nth vector
  thrust::device_vector<size_t> indices(n), col_vector(n);
  // To generate the indices, just to a counting vector from 0 to n, then add colnumber
  thrust::sequence(indices.begin(), indices.end());
  thrust::constant_iterator<size_t> colnumber_iter(colnumber);
  thrust::transform(indices.begin(),
                    indices.end(),
                    colnumber_iter.begin(),
                    indices.begin(),
                    thrust::add<size_t>);

  // Now gather them
  thrust::gather(indices.begin(), indices.end(), matrix.begin() col_vector.begin());

  return col_vector;
}

QR QR_factorization_GM(const std::vector<double>& A, const size_t n)
{
  assert(A.size() == n*n);
  std::vector<double> Q(n*n);
  std::vector<double> R(n*n, 0);

}

QR QR_factorization_GM_modified(const std::vector<double>& A, const size_t n)
{
  assert(A.size() == n*n);
  std::vector<double> Q(n*n);
  std::vector<double> R(n*n, 0);

}

QR QR_factorization_Householder(const std::vector<double>& A, const size_t n)
{
  assert(A.size() == n*n);
  std::vector<double> Q(n*n);
  std::vector<double> R(n*n, 0);

}
