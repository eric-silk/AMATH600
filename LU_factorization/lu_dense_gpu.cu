#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <math.h>
#include <assert.h>

struct LU
{
  LU() = default;
  std::vector<double> L;
  std::vector<double> U;
  size_t n;
};

// Borrowed from the transformations example code
// https://docs.nvidia.com/cuda/thrust/index.html#transformations
struct daxpy_functor
{
  const double a;
  daxpy_functor(double _a) : a(_a) {}
  __host__ __device__
    double operator()(const double& x, const double&y) const
    {
      return a * x + y;
    }
};

void daxpy(double A, thrust::device_vector<double>& X, thrust::device_vector<double>& Y)
{
  thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), daxpy_functor(A));
}

struct to_zero_functor
{
  const double epsilon;
  to_zero_functor(double _epsilon) : epsilon(_epsilon) {}
  __host__ __device__
    double operator()(const double& x) const
    {
      if (std::abs(x) <= epsilon)
        return 0;
      else
        return x;
    }
};

void to_zero(double epsilon, thrust::device_vector<double>& X)
{
  thrust::transform(X.begin(), X.end(), X.begin(), to_zero_functor(epsilon));
}

LU LU_factorization(const std::vector<double>& A, size_t n);
void print_matrix(const std::vector<double>& A, size_t n);
double rand_0_1(void);

int main(int argc, char **argv)
{
  int n = 0;
  if (argc >= 2)
  {
    n = atoi(argv[1]);
  }
  else
  {
    n = 1024;
  }

  std::vector<double> A;
  A.resize(n*n);
  std::srand(std::time(nullptr));
  std::generate(A.begin(), A.end(), rand_0_1);

  auto factored = LU_factorization(A, n);

  return 0;
}
double rand_0_1(void)
{
  return ((double) rand() / (RAND_MAX));
}

void print_matrix(const std::vector<double>& A, size_t n)
{
  for (size_t row = 0; row < n; ++row)
  {
    for (size_t col = 0; col < n; ++col)
    {
      std::cout << A[n*row+col] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

LU LU_factorization(const std::vector<double>& A, const size_t n)
{
  assert(A.size() == n*n);
  std::vector<double> U(n*n);
  std::vector<double> L(n*n, 0);

  // Initialize them
  std::copy(A.begin(), A.end(), U.begin());
  for (size_t i = 0; i < n; ++i)
  {
    L[i*(n+1)] = 1.0;
  }

  thrust::host_vector<double> top_row_host(n);
  thrust::device_vector<double> top_row_dev(n);
  thrust::host_vector<double> reducing_row_host(n);
  thrust::device_vector<double> reducing_row_dev(n);

  for (size_t col = 0; col < n-1; ++col)
  {
    std::copy(U.begin()+(col*n), U.begin()+((col+1)*n), top_row_host.begin());
    top_row_dev = top_row_host;
    for (int row = col+1; row < n; ++row)
    {
      size_t num_coeff = row*n+col;
      size_t den_coeff = col*n+col;
      double coeff = -(U[num_coeff] / U[den_coeff]);
      L[num_coeff] = coeff;

      // Copy the Rows to the host vector, then device vector
      size_t start_loc = row*n+col;
      size_t end_loc = start_loc+(n-col);
      thrust::fill(reducing_row_host.begin(), reducing_row_host.begin()+col, 0);
      std::copy(U.begin()+(start_loc),
                U.begin()+(end_loc),
                reducing_row_host.begin()+col);
      reducing_row_dev = reducing_row_host;
      // Scale and add
      daxpy(coeff, top_row_dev, reducing_row_dev);

      reducing_row_host = reducing_row_dev;
      thrust::copy(reducing_row_host.begin()+col,
                   reducing_row_host.end(),
                   U.begin()+start_loc);
    }
    //print_matrix(U, n);
  }

  // now round down the zeros given some threshold
  thrust::host_vector<double> final_host(n*n);
  std::copy(U.begin(), U.end(), final_host.begin());
  thrust::device_vector<double> final_dev = final_host;
  to_zero(1e-12, final_dev); 
  final_host = final_dev;
  thrust::copy(final_host.begin(), final_host.end(), U.begin());

  LU retval;
  retval.U = U;
  retval.L = L;

  return retval;
}
